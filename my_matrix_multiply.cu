#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "my_matmul.h"

__global__ void matmul_kernel(float* lhs, int lhs_rows, int lhs_cols,
	    float* rhs, int rhs_rows, int rhs_cols,
	    float* result) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;

  if (i >= lhs_rows * rhs_cols) {
    return;
  }

  
  int result_row = i / rhs_cols;
  int result_col = i % rhs_cols;

  float value = 0;
  for (int j = 0; j != lhs_cols; ++j) {
    float lhs_factor = lhs[result_row * lhs_cols + j];
    float rhs_factor = rhs[j * rhs_cols + result_col];
    value +=  lhs_factor * rhs_factor;
  }
  
  result[i] = value;
}


void matmul_cpu(float* lhs, int lhs_rows, int lhs_cols,
	    float* rhs, int rhs_rows, int rhs_cols,
                     float* result) {
  printf("kernel started\n");
  float* d_lhs = get_cuda_matrix(lhs_rows, lhs_cols, 0);
  float* d_rhs = get_cuda_matrix(rhs_rows, rhs_cols, 0);
  float* d_result = get_cuda_matrix(lhs_rows, rhs_cols, 0);

  hipMemcpy(d_lhs, lhs, lhs_rows * lhs_cols * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(d_rhs, rhs, rhs_rows * rhs_cols * sizeof(float),
             hipMemcpyHostToDevice);

  matmul(d_lhs, lhs_rows, lhs_cols, d_rhs, rhs_rows, rhs_cols, d_result);

  hipMemcpy(result, d_result, lhs_rows * rhs_cols * sizeof(float),
             hipMemcpyDeviceToHost);

  hipFree(d_lhs);
  hipFree(d_rhs);
  hipFree(d_result);

  printf("kernel finished\n");
}

void matmul(float* lhs, int lhs_rows, int lhs_cols,
	    float* rhs, int rhs_rows, int rhs_cols,
	    float* result) {
  
  int threadsPerBlock = 256;
  int num_elements = lhs_rows * rhs_cols;
  int blocksPerGrid = (num_elements / threadsPerBlock) + 1;
  matmul_kernel<<<blocksPerGrid, threadsPerBlock>>>(lhs, lhs_rows, lhs_cols,
						    rhs, rhs_rows, rhs_cols,
						    result);
  hipError_t err = hipSuccess;  
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error!\n");
  } else {
    ;//printf("Success!\n");
  }
}  
  

__global__ void initialize_kernel(float* matrix, int num_elements, float value) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i >= num_elements) {
    return;
  }

  matrix[i] = value;
}



void initialize(float* matrix, int num_elements, float value) {
  int threadsPerBlock = 256;
  int blocksPerGrid = (num_elements / threadsPerBlock) + 1;
  initialize_kernel<<<blocksPerGrid, threadsPerBlock>>>(matrix, num_elements, value);
  hipError_t err = hipSuccess;  
  err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error!\n");
  } else {
    ;//printf("Success!\n");
  }
}

float* get_cuda_matrix(int rows, int cols, float init_value) {
  size_t size = rows * cols * sizeof(float);
  float* d_A = NULL;
  hipError_t err = hipSuccess;  
  err = hipMalloc((void **)&d_A, size);

  if (err != hipSuccess) {
    printf("error!\n");
  } else {
    ;//printf("Success!\n");
  }

  initialize(d_A, rows * cols, init_value);
  return d_A;
}

void print_matrix(float* matrix, int rows, int cols) {
  for (int i = 0; i < rows; ++i) {
    for (int j = 0; j < cols; ++j) {
      printf("%f ", matrix[i * cols + j]);
    }
    printf("\n");
  }
}


void print_cpu(float* matrix, int rows, int cols) {
  size_t size = rows * cols * sizeof(float);
  float* host_matrix = (float*) malloc(size);

  hipMemcpy(host_matrix, matrix, size, hipMemcpyDeviceToHost);
  print_matrix(host_matrix, rows, cols);
  free(host_matrix);
}  
  
